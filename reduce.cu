
#include <hip/hip_runtime.h>
__global__ void reduce(float* input, float* output, int N){
    int allidx = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = threadIdx.x;
    int len = 256;
    __shared__ float  blockdata[256];
    blockdata[idx] = input[allidx];
    __syncthreads();
    while(len > 1){
        if(idx < len / 2){
            blockdata[idx] = blockdata[idx] + blockdata[idx + len / 2];
        }
        __syncthreads();
        len = len /2;
    }
    if(idx == 0){
        output[blockIdx.x] = blockdata[0];
    }
    return;
}
