#include "hip/hip_runtime.h"
__global__ void gmme(float* A, float* B, float* C, int M,int  N){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    int once = blockDim.x * blockDim.y * gridDim.x * gridDim.y
    int cnt = M * N / once;
    __shared__ float blockdataA[256];
    __shared__ float blockdataB[256];
    while(cnt--){
        blockdataA[idy * N + idx] = A[row * N + col];
        blockdataB[idy * N + idx] = B[row * N + col];
        __syncthreads();
        for(int i = 0; i < N; i++){
            C[idy * N + idx] += blockdataA[idy * N + i] * B[i * N + idx];
        }
    }
    __syncthreads();
    return;
}
