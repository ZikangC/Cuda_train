
#include <hip/hip_runtime.h>
#define blocksize  32
__global__ void gmme(float* A, float* B, float* C, int N){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    const int n = blocksize;
    __shared__ float blockdataA[n*n];
    __shared__ float blockdataB[n*n];
    float sum = 0.0;
    for(int i = 0; i < N/n; i++){
        blockdataA[idy*n + idx] = A[row*N + i*n + idx];
        blockdataB[idy*n + idx] = B[(idy + i*n) * N + col];
        __syncthreads();
        for(int j = 0; j < n; j++){
            sum += blockdataA[idy * n + j] * blockdataB[j * n + idx];
        }
        __syncthreads();
        
    }
    C[row * N + col] = sum;
    return;
}
